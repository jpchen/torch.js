
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorExtraJS.cu"
#else

// #include "THCTensorExtraJS.cuh"

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)

// #if defined(THC_REAL_IS_HALF)
//   #define con(val) __float2half(val)
// #el
#if defined(THC_REAL_IS_DOUBLE)
  #define con(val) val
#elif defined(THC_REAL_IS_FLOAT)
  #define con(val) val##f
#endif

// #define IMPLEMENT_OP(NAME, CFUNC, CONFUNC, REAL)             \
//   struct Tensor_##NAME##_##REAL##_Op {                                  \
//     __device__ __forceinline__ void operator()(real* dx, real* thisdx, real* x, real*y) const { \
//       *dx += (CFUNC)*(*thisdx);                                                \
//     }\
//     __device__ __forceinline__ void operator()(real* dx, real* thisdx, real* x) const { \
//       *dx += (CFUNC)*(*thisdx);                                                \
//     }\
//     __device__ __forceinline__ void operator()(real* dx, real* thisdx) const { \
//       *dx += (CFUNC)*(*thisdx);                                                \
//     }\
//                                                                        \
//   };
// #define IMPLEMENT_OP_WITH_CONST(NAME, CFUNC, CONFUNC, REAL)             \
//   struct Tensor_##NAME##_##REAL##_Op {                                  \
//     __device__ __forceinline__ void operator()(real* dx, real* thisdx, real* x, real*y) const { \
//       *dx += (CFUNC)*(*thisdx);                                                \
//     }\
//     __device__ __forceinline__ void operator()(real* dx, real* thisdx, real* x, real y) const { \
//       *dx += (CONFUNC)*(*thisdx);                                                \
//     }\
//     __device__ __forceinline__ void operator()(real* dx, real* thisdx, real* x) const { \
//       *dx += (CFUNC)*(*thisdx);                                                \
//     }\
//     __device__ __forceinline__ void operator()(real* dx, real* thisdx, real x) const { \
//       *dx += (CONFUNC)*(*thisdx);                                                \
//     }\
//     __device__ __forceinline__ void operator()(real* dx, real* thisdx) const { \
//       *dx += (CFUNC)*(*thisdx);                                                \
//     }\
//                                                                        \
//   };

#if defined(THC_REAL_IS_DOUBLE)
  #define powfct pow
#elif defined(THC_REAL_IS_FLOAT)
  #define powfct powf
#endif

#define sqrtfct THCNumerics<real>::sqrt
#define logfct THCNumerics<real>::log
#define cosfct THCNumerics<real>::cos
#define sinfct THCNumerics<real>::sin
#define sinhfct THCNumerics<real>::sinh
#define coshfct THCNumerics<real>::cosh


// d.neg = makeUnaryDerivatives('-1');
// this creates the op which kill get called within the kernel
//
IMPLEMENT_OP_DX_ARG0(dx_neg, con(-1.0), Real)
// we create a function called dx_neg which called dx_neg_Real_OP() created above
IMPLEMENT_DX_ARG0(neg, dx_neg, Real)

// d.add = makeBinaryDerivatives('1', '1');
IMPLEMENT_OP_DX_ARG0(dx_add, con(1.0), Real)

IMPLEMENT_DX_ARG0(add_f1, dx_add, Real)
IMPLEMENT_DX_ARG0_ACC(add_f1, dx_add, Real)

IMPLEMENT_DX_ARG0(add_f2, dx_add, Real)
IMPLEMENT_DX_ARG0_ACC(add_f2, dx_add, Real)

// // d.sub = makeBinaryDerivatives('1', '-1');
IMPLEMENT_DX_ARG0(sub_f1, dx_add, Real)
IMPLEMENT_DX_ARG0_ACC(sub_f1, dx_add, Real)

IMPLEMENT_DX_ARG0(sub_f2, dx_neg, Real)
IMPLEMENT_DX_ARG0_ACC(sub_f2, dx_neg, Real)

// d.mul = makeBinaryDerivatives('y', 'x');
IMPLEMENT_OP_DX_ARG1(dx_ident, *x, x, Real)
// simple function, send in our new dx_ident op
IMPLEMENT_DX_ARG1(mul_f1, dx_ident, Real)

// x, y same thing for mul, use the same op, and same setup
IMPLEMENT_DX_ARG1(mul_f2, dx_ident, Real)

// d.div = makeBinaryDerivatives('1/y', '-x/(y*y)');
IMPLEMENT_OP_DX_ARG1(dx_inv, con(1.0) / (*x), con(1.0) / x, Real)

IMPLEMENT_DX_ARG1(div_f1, dx_inv, Real)

// we need to do a regular version, and a version where the second input is constant
IMPLEMENT_OP_DX_ARG2(dx_x_y2_inv, -(*x)/((*y)*(*y)), -(*x) / (y*y), Real)

// use it as you'd expect here --
IMPLEMENT_DX_ARG2(div_f2, dx_x_y2_inv, Real)

// d.sqrt = makeUnaryDerivatives('1/(2*out)');
IMPLEMENT_OP_DX_ARG1(dx_div_2x, con(1.0) / (con(2.0) * (*x)), con(1.0) / (con(2.0) * (x)), Real)
// then we call that op
IMPLEMENT_DX_ARG1(sqrt, dx_div_2x, Real)
// d.exp = makeUnaryDerivatives('out');
// same as the dx_mul op
IMPLEMENT_DX_ARG1(exp, dx_ident, Real)
// d.log = makeUnaryDerivatives('1/x');
// same op as first input of div
IMPLEMENT_DX_ARG1(log,  dx_inv, Real)

// d.pow = makeBinaryDerivatives('y*Math.pow(x,y-1)', 'Math.log(x)*out');
IMPLEMENT_OP_DX_ARG2(dx_pow1, (*y) * powfct((*x), (*y) -1), y * powfct((*x), y -1),  Real)
IMPLEMENT_OP_DX_ARG2(dx_pow2, logfct(*x)*(*y), logfct(*x) * y,  Real)

IMPLEMENT_DX_ARG2(pow_f1, dx_pow1, Real)
IMPLEMENT_DX_ARG2(pow_f2, dx_pow2, Real)

// d.sin = makeUnaryDerivatives('Math.cos(x)');
IMPLEMENT_OP_DX_ARG1(dx_cos, cosfct(*x), cosfct(x),  Real)
IMPLEMENT_DX_ARG1(sin, dx_cos, Real)
// IMPLEMENT_THDX_1ARG(sin, cos(*x_data))

// d.cos = makeUnaryDerivatives('-Math.sin(x)');
IMPLEMENT_OP_DX_ARG1(dx_sin, -sinfct(*x), -sinfct(x),  Real)
IMPLEMENT_DX_ARG1(cos, dx_sin, Real)
// IMPLEMENT_THDX_1ARG(cos, -sin(*x_data))

// d.tan = makeUnaryDerivatives('1 + out*out');
IMPLEMENT_OP_DX_ARG1(dx_1sqr, con(1.0) + (*x) * (*x), con(1.0) + (x*x),  Real)
IMPLEMENT_DX_ARG1(tan, dx_1sqr, Real)
// IMPLEMENT_THDX_1ARG(tan, 1 + (*x_data)*(*x_data))

// d.asin = makeUnaryDerivatives('1 / Math.sqrt(1 - x*x)');
IMPLEMENT_OP_DX_ARG1(dx_1sqrt, con(1.0) / sqrtfct(con(1.0) - (*x) * (*x)), con(1.0) / sqrtfct(con(1.0) - (x * x)), Real)

// IMPLEMENT_THDX_1ARG(asin, 1 / sqrt(1 - (*x_data)*(*x_data)))
IMPLEMENT_DX_ARG1(asin, dx_1sqrt, Real)
// d.acos = makeUnaryDerivatives('-1 / Math.sqrt(1 - x*x)');
// IMPLEMENT_THDX_1ARG(acos, -1 / sqrt(1 - (*x_data)*(*x_data)))
IMPLEMENT_OP_DX_ARG1(dx_neg1sqrt, con(-1.0) / sqrtfct(con(1.0) - (*x) * (*x)), con(-1.0) / sqrtfct(con(1.0) - (x * x)), Real)
IMPLEMENT_DX_ARG1(acos, dx_neg1sqrt, Real)
// d.atan = makeUnaryDerivatives('1 / (1 + x*x)');
// IMPLEMENT_THDX_1ARG(atan, 1 / (1 + (*x_data)*(*x_data)))
IMPLEMENT_OP_DX_ARG1(dx_inv1plussqr, con(1.0) / (con(1.0) + (*x) * (*x)), con(1.0) / (con(1.0) + (x * x)), Real)
IMPLEMENT_DX_ARG1(atan, dx_inv1plussqr, Real)

// d.atan2 = makeBinaryDerivatives('y/(x*x + y*y)', '-x/(x*x + y*y)');
IMPLEMENT_OP_DX_ARG2(dx_atan2f1, (*y) / ((*x)*(*x) + (*y)*(*y)), y / ((*x)*(*x) + (y * y)), Real)
IMPLEMENT_OP_DX_ARG2(dx_atan2f2, -(*x) / ((*x)*(*x) + (*y)*(*y)), -(*x) / ((*x)*(*x) + (y * y)), Real)

IMPLEMENT_DX_ARG2(atan2_f1, dx_atan2f1, Real)
IMPLEMENT_DX_ARG2(atan2_f2, dx_atan2f2, Real)
// IMPLEMENT_THDX_2ARG(atan2_f1, (*y_data)/((*x_data)*(*x_data) + (*y_data)*(*y_data)))
// IMPLEMENT_THDX_2ARG(atan2_f2, -(*x_data)/((*x_data)*(*x_data) + (*y_data)*(*y_data)))

// d.sinh = makeUnaryDerivatives('Math.cosh(x)');
IMPLEMENT_OP_DX_ARG1(dx_cosh, coshfct(*x), coshfct(x), Real)
// IMPLEMENT_THDX_1ARG(sinh, cosh(*x_data))
IMPLEMENT_DX_ARG1(sinh, dx_cosh, Real)

// d.cosh = makeUnaryDerivatives('Math.sinh(x)');
IMPLEMENT_OP_DX_ARG1(dx_sinh, sinhfct(*x), sinhfct(x), Real)
// IMPLEMENT_THDX_1ARG(cosh, sinh(*x_data))
IMPLEMENT_DX_ARG1(cosh, dx_sinh, Real)

// d.tanh = makeUnaryDerivatives('1 - out*out');
IMPLEMENT_OP_DX_ARG1(dx_1minsqr, con(1.0) - (*x)*(*x), con(1.0) - (x * x), Real)
IMPLEMENT_DX_ARG1(tanh, dx_1minsqr, Real)
// IMPLEMENT_THDX_1ARG(tanh, 1 - (*x_data)*(*x_data))

// d.asinh = makeUnaryDerivatives('1 / Math.sqrt(x*x + 1)');
IMPLEMENT_OP_DX_ARG1(dx_isqrtplus, con(1.0) / sqrtfct(con(1.0) + (*x) * (*x)), con(1.0) / sqrtfct(con(1.0) + (x * x)), Real)
IMPLEMENT_DX_ARG1(asinh, dx_isqrtplus, Real)
// IMPLEMENT_THDX_1ARG(asinh, 1 / sqrt((*x_data)*(*x_data) + 1))

// d.acosh = makeUnaryDerivatives('1 / Math.sqrt(x*x - 1)');
IMPLEMENT_OP_DX_ARG1(dx_isqrtmin, con(1.0) / sqrtfct((*x) * (*x) - con(1.0)), con(1.0) / sqrtfct((x * x) - con(1.0)), Real)
IMPLEMENT_DX_ARG1(acosh, dx_isqrtmin, Real)
// IMPLEMENT_THDX_1ARG(acosh, 1 / sqrt((*x_data)*(*x_data) - 1))

// d.atanh = makeUnaryDerivatives('1 / (1 - x*x)');
IMPLEMENT_OP_DX_ARG1(dx_inv1minsqr, con(1.0) / (con(1.0) - (*x) * (*x)), con(1.0) / (con(1.0) - (x * x)), Real)
// IMPLEMENT_THDX_1ARG(atanh, 1 / (1 - (*x_data)*(*x_data)))
IMPLEMENT_DX_ARG1(atanh, dx_inv1minsqr, Real)

// d.sigmoid = makeUnaryDerivatives('out * (1 - out)');
IMPLEMENT_OP_DX_ARG1(dx_sigop, (*x) * (con(1.0) - (*x)), x * (con(1.0) - x),  Real)
IMPLEMENT_DX_ARG1(sigmoid, dx_sigop, Real)
// IMPLEMENT_THDX_1ARG(sigmoid, (*x_data) * (1 - (*x_data)))


// IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  log, THCNumerics<real>::log,   Real)
// IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, THCNumerics<real>::log1p, Real)

#undef con
#undef powfct
#undef logfct
#undef sqrtfct

#undef sinfct
#undef cosfct

#undef coshfct
#undef sinhfct
// remove the helper for different constant types (float/double)

#endif
//close out if float/double

#endif
