
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathPointwise.cu"
#else

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)             \
  struct Tensor_##NAME##_##REAL##_Op {                                  \
    __device__ __forceinline__ void operator()(real* out, real* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(real* v) const {         \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCTensor_(NAME)(THCState* state, THCTensor* self_, THCTensor* src) { \
    THAssert(THCTensor_(checkGPU)(state, 2, self_, src));               \
    if (self_ == src) {                                                 \
      if (!THC_pointwiseApply1(state, self_, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    } else {                                                            \
      THCTensor_(resizeAs)(state, self_, src);                          \
                                                                        \
      if (!THC_pointwiseApply2(state, self_, src, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(cudaGetLastError());                                    \
  }

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC, REAL) \
  IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  log, THCNumerics<real>::log,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, THCNumerics<real>::log1p, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  exp, THCNumerics<real>::exp,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cos, THCNumerics<real>::cos,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sin, THCNumerics<real>::sin,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sqrt, THCNumerics<real>::sqrt,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(rsqrt, THCNumerics<real>::rsqrt, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( ceil, THCNumerics<real>::ceil,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(floor, THCNumerics<real>::floor, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(trunc, THCNumerics<real>::trunc, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  neg, THCNumerics<real>::neg,   Real)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( acos, THCNumerics<real>::acos,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( cosh, THCNumerics<real>::cosh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( asin, THCNumerics<real>::asin,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sinh, THCNumerics<real>::sinh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  tan, THCNumerics<real>::tan,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( atan, THCNumerics<real>::atan,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( tanh, THCNumerics<real>::tanh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(round, THCNumerics<real>::round, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( frac, THCNumerics<real>::frac,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( cinv, THCNumerics<real>::cinv,  Real)

#endif

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  abs, THCNumerics<real>::abs,   Real)

#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

void THCTensor_(sign)(THCState* state, THCTensor* self_, THCTensor* src) {
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorSignOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorSignOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(sigmoid)(THCState* state, THCTensor* self_, THCTensor* src) {
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorSigmoidOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorSigmoidOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

#endif

THC_API void
THCTensor_(cadd)(THCState *state, THCTensor *self_, THCTensor* src1, real value, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == ScalarConvert<int, real>::to(1)) {
      // self += src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorAddOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += value * src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorCAddOp<real>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    if (value == ScalarConvert<int, real>::to(1)) {
      // self = src1 + src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorAddOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 + value * src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorCAddOp<real>(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(csub)(THCState *state, THCTensor *self_, THCTensor* src1, real value, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == ScalarConvert<int, real>::to(1)) {
      // self -= src2
      if (!THC_pointwiseApply2(state, self_, src2, TensorSubOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += -value * src2
      if (!THC_pointwiseApply2(state, self_, src2,
                                   TensorCAddOp<real>(
                                     ScalarNegate<real>::to(value)))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    if (value == ScalarConvert<int, real>::to(1)) {
      // self = src1 - src2
      if (!THC_pointwiseApply3(state, self_, src1, src2, TensorSubOp<real>())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 - value * src2
      if (!THC_pointwiseApply3(state, self_, src1, src2,
                                   TensorCAddOp<real>(
                                     ScalarNegate<real>::to(value)))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cmul)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self *= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorMulOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 * src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorMulOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cpow)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self = pow(self, src2)
    if (!THC_pointwiseApply2(state, self_, src2, TensorCPowOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = pow(src1, src2)
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorCPowOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(cdiv)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self *= src2
    if (!THC_pointwiseApply2(state, self_, src2, TensorDivOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 * src2
    if (!THC_pointwiseApply3(state, self_, src1, src2, TensorDivOp<real>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

#endif
